#include "hip/hip_runtime.h"
/*
 *     This file is part of Common_GPU_Operations.
 *     Copyright (C) 2009-2021  Evstigneev Nikolay Mikhaylovitch <evstigneevnm@ya.ru>, Ryabkov Oleg Igorevitch
 *
 *     This program is free software: you can redistribute it and/or modify
 *     it under the terms of the GNU General Public License as published by
 *     the Free Software Foundation, either version 3 of the License, or
 *     (at your option) any later version.
 *
 *     This program is distributed in the hope that it will be useful,
 *     but WITHOUT ANY WARRANTY; without even the implied warranty of
 *     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *     GNU General Public License for more details.
 *
 *     You should have received a copy of the GNU General Public License
 *     along with this program.  If not, see <https://www.gnu.org/licenses/>.
 *      */

// This file is part of SimpleCFD.

// SimpleCFD is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, version 2 only of the License.

// SimpleCFD is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.

// You should have received a copy of the GNU General Public License
// along with SimpleCFD.  If not, see <http://www.gnu.org/licenses/>.

#include <cstdio>
#include <utils/init_cuda.h>
#include <utils/constant_data.h>

struct t_test
{
    int x;
};

void test_init();
void test_test();

DEFINE_CONSTANT_BUFFER(t_test, buf)

__global__ void ker_test()
{
    printf("device test1: buf().x = %d\n", buf().x);
}

int main()
{
    t_test  test;
    test.x = 152;

    utils::init_cuda(0);

    COPY_TO_CONSTANT_BUFFER(buf, test);
    test_init();

    printf("host test1: buf().x = %d\n", buf().x);
    ker_test<<<1,1>>>();
    hipDeviceSynchronize();

    test_test();

    return 0;
}